#include <iostream>
#include <cstdio>
#include <time.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

using namespace std;

void Read(int** A, int** B, int *la, int *lb, const char *filename) {    
	FILE *fp;
	fp = fopen(filename, "r");
  	fscanf(fp, "%d %d\n", la, lb);

	int* Atemp = new int[(*la) * 2];
	int* Btemp = new int[(*lb) * 2];

	for (int i = 0; i < (*la); i++){
		fscanf(fp, "%d %d\n", &(Atemp[2*i]), &(Atemp[2*i + 1]));
	}

	for (int j = 0; j < (*lb); j++){
		fscanf(fp, "%d %d\n", &(Btemp[2*j]), &(Btemp[2*j + 1]));
	}

	*A = Atemp;
	*B = Btemp;
}

void ReadSoA(int** A, int** B, int *la, int *lb, const char *filename) {    
	FILE *fp;
	fp = fopen(filename, "r");
  	fscanf(fp, "%d %d\n", la, lb);

	int* Atemp = new int[(*la) * 2];
	int* Btemp = new int[(*lb) * 2];

	for (int i = 0; i < (*la); i++){
		fscanf(fp, "%d %d\n", &(Atemp[i]), &(Atemp[*la + i]));
	}

	for (int j = 0; j < (*lb); j++){
		fscanf(fp, "%d %d\n", &(Btemp[j]), &(Btemp[*lb + j]));
	}

	*A = Atemp;
	*B = Btemp;
}

void Write(int* intersecciones, int la, int lb, const char *filename) {
	FILE *fp;
	fp = fopen(filename, "w");

	for (int i = 0; i < (la*lb*2); i++){
		if (i%2 == 0){
			if ((i != 0) && (intersecciones[i] == 0) && (intersecciones[i+1] == 0)){
				break;
			}
			fprintf(fp, "%d %d\n", intersecciones[i], intersecciones[i + 1]);
		}
	}
	fclose(fp);
}

void WriteHash(int* intersecciones, int la, int lb, const char *filename) {
	FILE *fp;
	fp = fopen(filename, "w");

	for (int i = 0; i < la; i++){
		for (int j = 0; j < lb; j++){
			if (intersecciones[2*lb*i + 2*j] == -1){
				break;
			}
			fprintf(fp, "%d %d\n", intersecciones[2*lb*i + 2*j], intersecciones[2*lb*i + 2*j + 1]);
		}

	}

	fclose(fp);
}

bool seIntersecta(int aStart, int aEnd, int bStart, int bEnd){

	if ((aEnd < bStart) || (bEnd < aStart)){
		return false;
	}
	else{
		return true;
	}
}

void interseccionConjuntos(int* A, int *B,int *intersecciones, 
						int la, int lb){		
	
	int aStart, aEnd, bStart, bEnd;
	int posicion = 0;
	
	for (int i = 0; i < la; i++){
		aStart = A[2*i];
		aEnd = A[2*i + 1];

		for (int j = 0; j < lb; j++){
			bStart = B[2*j];
			bEnd = B[2*j + 1];

			if (seIntersecta(aStart, aEnd, bStart, bEnd)){
				//Guardo el número del intervalo (partiendo de 0)
				intersecciones[posicion] = i;
				intersecciones[posicion + 1] = j;
				posicion += 2;
			}
		}
	}
}


//Buscar el indice del intervalo de B que termina antes de que sStart inicie.
__device__ void binarySearchEnds(int *B, int lB, int aStart, int *slice){
	int low = 0;
	int high = lB - 1;

	while(low <= high){
		int mid = (low + high)/2;

		if (B[2*mid + 1] >= aStart){
			high = mid - 1;
		} else { // mid > target
			low = mid + 1;
		}
	}
	slice[0] = high;
}

//Deberia ser correcto, buscar el elemento en B, que inicia despues de que sEnd termina.
//O sea, el siguiente numero mayor a sEnd.
__device__ void binarySearchStart(int *B, int lB, int sEnd, int *slice){
	int low = 0;
	int high = lB - 1;

	while(low <= high){
		int mid = (low + high)/2;
		if (B[2*mid] <= sEnd){
			low = mid + 1;
		} else { // mid > target
			high = mid - 1;
		}
	}
	slice[1] = low;
}

__device__ bool isAnIntersect(int aStart, int aEnd, int bStart, int bEnd){
	if ((aEnd < bStart) || (bEnd < aStart)){
		return false;
	}
	else{
		return true;
	}
}


__global__ void setIntersection_Kernel2(int *A, int *B, int lA, int lB, int *intercepts, int *lenIntercepts){
	int Id = threadIdx.x + blockIdx.x * blockDim.x;

	if (Id >= lA) return;

	int* slice = new int[2];
	int aStart = A[2*Id];
	int aEnd = A[2*Id + 1];

	//Cortamos B segun el a_Id. Via busqueda binarias
	binarySearchEnds(B, lB, aStart, slice);
	binarySearchStart(B, lB, aEnd, slice);

	if (slice[0] > slice[1]){
		//No hay interseccion.
		return;
	}

	int *tempInter = new int[2*(slice[1] - slice[0])];  
	int tempInterFounds = 0;
	int bStart, bEnd;
	
	//Retornamos los intervalos que se intersectan dentro de las slices.
	for (int i = slice[0]; i <= slice[1]; i++){
		bStart = B[2*i];
		bEnd = B[2*i + 1];
		if (isAnIntersect(aStart, aEnd, bStart, bEnd)){
			//tempInter[2 * tempInterFounds] = Id;
			//tempInter[2 * tempInterFounds + 1] = i; 
			intercepts[Id * 2 * lB + 2*tempInterFounds] = Id; 
			intercepts[Id * 2 * lB + 2*tempInterFounds + 1] = i;

			tempInterFounds += 1;
		}
	}



}

int main(int argc, char **argv){

	// Largo del arreglo A y B, respectivamente.
	int la, lb;
	// Conjuntos de intervalos A y B.
	int *A, *B;
	int *intersecciones;
	clock_t t1, t2;
	
	char filename[] = {"input.txt\0"};
	char outputFilename[] = {"output.txt\0"};

	Read(&A, &B, &la, &lb, filename); 

	// for (int i = 0; i < la*2; i++){
	// 	std::cout << A[i] << std::endl;
	// }

	// Parte CPU

	intersecciones = new int[la*lb*2];

	t1 = clock();
	interseccionConjuntos(A, B, intersecciones, la, lb);
	t2 = clock();

	double ms = 1000.0 * (double)(t2 -t1) / CLOCKS_PER_SEC;

	std::cout << "Tiempo algoritmo en CPU = " << ms << "[ms]" << std::endl;

	Write(intersecciones, la, lb, outputFilename);

	delete[] intersecciones;

	//Kernel 2 - Binary Search + ...

	hipEvent_t ct1, ct2;
	int *Adev, *Bdev;
	int *interdev, *interhost;
	int *intercepts, *interceptsdev, *lenIntercepts, *lenInterceptsdev;
	lenIntercepts = 0;

    hipEventCreate(&ct1);
    hipEventCreate(&ct2);

    //KERNEL 1

    int gs, bs;
    hipMalloc((void**)&Adev, 2 * la * sizeof(int));
    hipMalloc((void**)&Bdev, 2 * lb * sizeof(int));

    hipMalloc((void**)&interceptsdev, 2 * la * lb * sizeof(int));
    hipMemset(interceptsdev, -1, (2* la* lb) * sizeof(int));

    hipMalloc((void**)&lenInterceptsdev, sizeof(int));

    hipMemcpy(Adev, A, 2 * la * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(Bdev, B, 2 * lb * sizeof(int), hipMemcpyHostToDevice); 
    hipMemcpy(lenInterceptsdev, lenIntercepts, sizeof(int), hipMemcpyHostToDevice); 

    bs = 256;
    gs = (int)ceil((float) la / bs);

    hipEventRecord(ct1);
    setIntersection_Kernel2<<<gs, bs>>>(Adev, Bdev, la, lb, interceptsdev, lenInterceptsdev);
    hipEventRecord(ct2);
    hipEventSynchronize(ct2);

    float dt;
    hipEventElapsedTime(&dt, ct1, ct2);

    intercepts = new int[2 * la * lb];
    hipMemcpy(intercepts, interceptsdev, (2 * la * lb) * sizeof(int), hipMemcpyDeviceToHost);

    printf("\nTiempo GPU 2 + Binary Searchs: %f[ms]\n", dt);
    
    WriteHash(intercepts, la, lb, "outputkernel2.txt");



	return 0;

}